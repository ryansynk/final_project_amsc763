#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

__global__ void print_ptr(double *A, int A_rows, int A_cols) {
    // Data stored in column-major order
    //
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < A_cols; j++) {
            printf("%f ", A[j*A_rows + i]);
        }
        printf("\n");
    }
}

__global__ void print_ptr_num(double *A) {
    // Data stored in column-major order
    printf("%f\n", *A);
}

static const char *_cudaGetErrorEnum(hipblasStatus_t error) {
    switch (error) {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
    }
    return "<unknown>";
}

// On-device householder transform
//__global__ void gpu_house(cublasHandle_t handle, int row_beg, int rows, int cols, double *d_x, double *d_v, double *beta, int col) {
//TODO get rid of cols parameter
__global__ void gpu_house(hipblasHandle_t handle, int k, int rows, int cols, double *d_x, double *d_v, double *beta) {
    if (d_v == NULL || d_x == NULL || beta == NULL) {
        return;
    }
    double norm_x = 0.0;
    *beta = 0.0;
    // Data stored in column-major order
    // d_v gets first column
    for (int i = 0; i < rows - k; i++) {
        (d_v + k)[i] = (d_x + k + k * rows)[i];
        norm_x += (d_x + k + k * rows)[i] * (d_x + k + k * rows)[i];
    }

    norm_x = sqrt(norm_x);
    if (norm_x == 0.0) {
        *beta = 0.0;
    } else {
        (d_v + k)[0] = (d_v + k)[0] - norm_x;

        for (int i = 0; i < rows - k; i++) {
            *beta += (d_v + k)[i] * (d_v + k)[i];
        }
        *beta = -2.0 / *beta;
    }
}

// extern "C"
//int gpu_qr(Matrix *A, Matrix *Q, Matrix *R, int A_rows, int A_cols) {
extern "C" int gpu_qr(double *A, double *Q, double *R, int A_rows, int A_cols) {
    int status = EXIT_SUCCESS;
    if (A == NULL || Q == NULL || R == NULL) {
        status = EXIT_FAILURE;
    } 

    int Q_rows = A_rows;
    int Q_cols = A_rows;
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < A_rows; j++) {
            if (i == j) {
                Q[i * A_rows + j] = 1.0;
            } else {
                Q[i * A_rows + j] = 0.0;
            }
        }
    }

    int R_rows = A_rows;
    int R_cols = A_cols;
    for (int i = 0; i < A_rows * A_cols; i++) {
        R[i] = A[i];
    }

    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("stat = %s\n", _cudaGetErrorEnum(stat));
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    double* dev_A = NULL;
    double* dev_Q = NULL;
    double* dev_R = NULL;
    double* dev_x = NULL;
    double* dev_v = NULL;
    double* dev_Rv = NULL;
    double* dev_Qv = NULL;
    double* dev_beta = NULL;
    double* dev_alpha = NULL;
    double* dev_gamma = NULL;

    double beta = 0.0;
    double alpha = 1.0;
    double gamma = 0.0;

    cudaStat = hipMalloc(&dev_A, A_rows * A_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_Q, Q_rows * Q_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_R, R_rows * R_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_x, R_rows * sizeof(double));
    cudaStat = hipMalloc(&dev_v, R_rows * sizeof(double));
    cudaStat = hipMalloc(&dev_Rv, R_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_Qv, Q_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_beta, sizeof(double));
    cudaStat = hipMalloc(&dev_alpha, sizeof(double));
    cudaStat = hipMalloc(&dev_gamma, sizeof(double));

    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed\n");
        return EXIT_FAILURE;
    }

    cudaStat = hipMemcpy(dev_A, A, A_rows * A_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_Q, Q, Q_rows * Q_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_R, R, R_rows * R_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_alpha, &alpha, sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_gamma, &gamma, sizeof(double), hipMemcpyHostToDevice);

    if (cudaStat != hipSuccess) {
        printf("host to device memory copy failed\n");
        return EXIT_FAILURE;
    }
    for (int k = 0; k < A_cols; k++) {
        gpu_house<<<1,1>>>(handle, k, R_rows, R_cols, dev_R, dev_v, dev_beta);
        cudaStat = hipMemcpy(&beta, dev_beta, sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        // Gets dev_Rv = R[k:m, k:n] @ v
        stat = hipblasDgemv(handle, HIPBLAS_OP_T, 
                           R_rows - k, R_cols - k, 
                           &alpha, 
                           (dev_R + k + k * R_rows), R_rows,
                           (dev_v + k), 1,
                           &gamma,
                           (dev_Rv + k), 1);

        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf("stat = %s\n", _cudaGetErrorEnum(stat));
            hipFree(dev_A);
            hipFree(dev_Q);
            hipFree(dev_R);
            hipFree(dev_x);
            hipFree(dev_v);
            hipFree(dev_Rv);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }

        // Sets R[k:m, k:n] = R[k:m, k:n] - beta * v @ dev_Rv.T
        stat = hipblasDger(handle, 
                          R_rows - k, R_cols - k,
                          &beta,
                          (dev_v + k), 1,
                          (dev_Rv + k), 1,
                          (dev_R + k + k * R_rows), R_rows);

        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf("stat = %s\n", _cudaGetErrorEnum(stat));
            hipFree(dev_A);
            hipFree(dev_Q);
            hipFree(dev_R);
            hipFree(dev_x);
            hipFree(dev_v);
            hipFree(dev_Rv);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }

        // Gets dev_Qv = Q[:, k:m] @ v
        stat = hipblasDgemv(handle, HIPBLAS_OP_N, 
                           Q_rows, Q_cols - k, 
                           &alpha, 
                           (dev_Q + k * Q_rows), Q_rows,
                           (dev_v + k), 1,
                           &gamma,
                           (dev_Qv + k), 1);

        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf("stat = %s\n", _cudaGetErrorEnum(stat));
            hipFree(dev_A);
            hipFree(dev_Q);
            hipFree(dev_R);
            hipFree(dev_x);
            hipFree(dev_v);
            hipFree(dev_Rv);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }
        printf("beta = %f\n", beta);

        // Sets Q[:, k:m] = Q[:, k:m] - beta * (Q[:, k:m] @ v) @ v.T
        stat = hipblasDger(handle, 
                          Q_rows, Q_cols - k,
                          &beta,
                          (dev_Qv + k), 1,
                          (dev_v + k), 1,
                          (dev_Q + k * Q_rows), Q_rows);

        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf("stat = %s\n", _cudaGetErrorEnum(stat));
            hipFree(dev_A);
            hipFree(dev_Q);
            hipFree(dev_R);
            hipFree(dev_x);
            hipFree(dev_v);
            hipFree(dev_Rv);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }
    }

    cudaStat = hipMemcpy(A, dev_A, A_rows * A_cols * sizeof(double), hipMemcpyDeviceToHost);
    cudaStat = hipMemcpy(Q, dev_Q, Q_rows * Q_cols * sizeof(double), hipMemcpyDeviceToHost);
    cudaStat = hipMemcpy(R, dev_R, R_rows * R_cols * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dev_A);
    hipFree(dev_Q);
    hipFree(dev_R);
    hipFree(dev_x);
    hipFree(dev_v);
    hipFree(dev_Rv);
    hipblasDestroy(handle);
    return status;
}


extern "C" int gpu_block_qr(double *A, double *Q, double *R, int A_rows, int A_cols, int r) {
    int status = EXIT_SUCCESS;
    if (A == NULL || Q == NULL || R == NULL) {
        status = EXIT_FAILURE;
    } 

    int Q_rows = A_rows;
    int Q_cols = A_rows;
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < A_rows; j++) {
            if (i == j) {
                Q[i * A_rows + j] = 1.0;
            } else {
                Q[i * A_rows + j] = 0.0;
            }
        }
    }

    int R_rows = A_rows;
    int R_cols = A_cols;
    for (int i = 0; i < A_rows * A_cols; i++) {
        R[i] = A[i];
    }

    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("stat = %s\n", _cudaGetErrorEnum(stat));
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    } 

    double* dev_A = NULL;
    double* dev_Q = NULL;
    double* dev_R = NULL;
    double* dev_x = NULL;
    //double* dev_v = NULL;
    double* dev_Rv = NULL;
    double* dev_Qv = NULL;
    double* dev_beta = NULL;
    double* dev_alpha = NULL;
    double* dev_gamma = NULL;

    double* dev_Y = NULL;
    double* dev_W = NULL;

    double* dev_B = NULL;
    double* dev_Vmat = NULL;
    double* B = NULL;

    double* dev_Yt_v = NULL;
    double* dev_WYt_v = NULL;

    // intermediate -- used for updating R
    double* dev_WTR = NULL;  // "W transpose times R" 
    double* dev_YWTR = NULL; // "Y times W transpose times R" 

    // intermediate -- used for updating Q
    double* dev_WYT = NULL;  // "W times Y transpose"
    double* dev_QWYT = NULL; // "Q times W times Y transpose"

    double beta = 0.0;
    double alpha = 1.0;
    double gamma = 0.0;
    int s = 0;
    int u = 0;

    cudaStat = hipMalloc(&dev_A, A_rows * A_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_Q, Q_rows * Q_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_R, R_rows * R_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_x, R_rows * sizeof(double));

    //cudaStat = cudaMalloc(&dev_v, R_rows * sizeof(double));

    cudaStat = hipMalloc(&dev_Rv, R_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_Qv, Q_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_beta, sizeof(double));
    cudaStat = hipMalloc(&dev_alpha, sizeof(double));
    cudaStat = hipMalloc(&dev_gamma, sizeof(double));

    cudaStat = hipMalloc(&dev_B, r * sizeof(double));
    cudaStat = hipMalloc(&dev_Vmat, A_rows * r * sizeof(double));

    cudaStat = hipMalloc(&dev_Y, A_rows * r * sizeof(double));
    cudaStat = hipMalloc(&dev_W, A_rows * r * sizeof(double));

    cudaStat = hipMalloc(&dev_Yt_v, r * sizeof(double));
    cudaStat = hipMalloc(&dev_WYt_v, A_rows * sizeof(double));

    cudaStat = hipMalloc(&dev_WTR, r * (R_cols - r) * sizeof(double));
    cudaStat = hipMalloc(&dev_YWTR, R_rows * (R_cols - r) * sizeof(double));

    cudaStat = hipMalloc(&dev_WYT, R_rows * R_rows * sizeof(double));
    cudaStat = hipMalloc(&dev_QWYT, Q_rows * R_rows * sizeof(double));

        // R[s:, s + r:] = R[s:, s + r:] + Y @ (W.T @ R[s:, s + r:])
        // gemm  dev_WTR <-- W.T @ (R + s + (s + r) * R_rows)
        // gemm  dev_YWTR <-- Y @ dev_WTR
        // axpy  R + s + (s + r) * R_rows <-- dev_YWTR + (R + s + (s + r) * R_rows)
        //
        // Update Q
        // Q[:, s:] = Q[:, s:] + Q[:, s:] @ W  @ Y.T
        // gemm dev_WYT <-- W @ Y.T
        // gemm dev_QWYT <-- (Q + s * Q_rows) @ dev_WYT
        // axpy (Q + s * Q_rows) <-- dev_QWYT + (Q + s * Q_rows)

        // R[s:, s + r:] is,  at max, R_rows x R_cols - r
        // W is R_rows x r, so W.T @ R[s:, s + r:] = dev_WTR is r x (R_cols - r)
        // Y is R_rows x r, so Y @ dev_WTR = dev_YWTR is R_rows x (R_cols - r)

        // W is R_rows x r, Y is R_rows x r, so W @ Y.T = dev_WYT is R_rows x R_rows
        // at max, Q[:, s:] is Q_rows x Q_rows, so Q[:, s:] @ (W @ Y.T) = dev_QWYT is (Q_rows x R_rows)

    // V is m x r, v is m x 1
    // Y is m x r, Y.T @ v = Yt_v is r x 1
    // W is m x r, W @ Yt_v = WYt_v is m x 1
    // z is m x 1

    // V is (m - s) x r, 

        // gemv Yt_v <-- Y.T @ v
        // gemm WYt_v <-- -B[j] * W @ Yt_v
        // gemm z <-- -B[j]v + WYt_v

    B = (double *)malloc(r * sizeof(double));

    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed\n");
        return EXIT_FAILURE;
    }

    cudaStat = hipMemcpy(dev_A, A, A_rows * A_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_Q, Q, Q_rows * Q_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_R, R, R_rows * R_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_alpha, &alpha, sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_gamma, &gamma, sizeof(double), hipMemcpyHostToDevice);

    if (cudaStat != hipSuccess) {
        printf("host to device memory copy failed\n");
        return EXIT_FAILURE;
    }

    for (int k = 0; k < (A_cols / r); k++) {
        printf("k = %d\n", k);
        s = k * r;
        //TODO: zero out dev_Vmat?
        for (int j = 0; j < r; j++) {

            //gpu_house<<<1,1>>>(handle, k, R_rows, R_cols, dev_R, dev_v, dev_beta);
            //cudaStat = cudaMemcpy(&beta, dev_beta, sizeof(double), cudaMemcpyDeviceToHost);
            //cudaDeviceSynchronize();

            //// Gets dev_Rv = R[k:m, k:n] @ v
            //stat = cublasDgemv(handle, CUBLAS_OP_T, 
            //                   R_rows - k, R_cols - k, 
            //                   &alpha, 
            //                   (dev_R + k + k * R_rows), R_rows,
            //                   (dev_v + k), 1,
            //                   &gamma,
            //                   (dev_Rv + k), 1);

            printf("j = %d\n", j);
            u = s + j;
            //gpu_house<<<1,1>>>(handle, u, R_rows, R_cols, dev_R, dev_Vmat + u * R_rows, dev_beta);
            printf("\n");
            printf("===== gpu_house =====\n");

            printf("R[u:, u]\n");
            print_ptr<<<1,1>>>(dev_R + u + u * R_rows, R_rows - u, 1);
            hipDeviceSynchronize();

            printf("u = %d\n", u);

            gpu_house<<<1,1>>>(handle, u, R_rows, R_cols, dev_R, (dev_Vmat + j * R_rows + j - u), dev_beta);
            cudaStat = hipMemcpy((B + j), dev_beta, sizeof(double), hipMemcpyDeviceToHost);

            printf("v = \n");
            print_ptr<<<1,1>>>(dev_Vmat + j * R_rows + j, R_rows - j, 1);
            hipDeviceSynchronize();
            printf("beta = %f\n", *(B + j));
            //cudaStat = cudaMemcpy(&beta, dev_beta, sizeof(double), cudaMemcpyDeviceToHost);
            //B[j] = beta;
            // R[u:m, u:(s + r)].T @ v

            printf("\n");
            printf("===== hipblasDgemv =====\n");

            printf("R = \n");
            print_ptr<<<1,1>>>(dev_R, R_rows, R_cols);
            hipDeviceSynchronize();

            printf("R[u:m, u:(s + r)] = \n");
            print_ptr<<<1,1>>>(dev_R + u + u * R_rows, R_rows - u, (s + r) - u);
            hipDeviceSynchronize();

            printf("dev_Vmat[:, j] = \n");
            print_ptr<<<1,1>>>(dev_Vmat + j + j * R_rows, R_rows - j, 1);
            hipDeviceSynchronize();

            stat = hipblasDgemv(handle, HIPBLAS_OP_T, 
                               R_rows - u, (s + r) - u, 
                               &alpha, 
                               (dev_R + u + u * R_rows), R_rows,
                               //(dev_Vmat + u + u * R_rows), 1,
                               (dev_Vmat + j + j * R_rows), 1,
                               &gamma,
                               (dev_Rv + u), 1);

            printf("R[u:m, u:(s + r)].T @ v\n");
            print_ptr<<<1,1>>>((dev_Rv + u), (s + r) - u, 1);
            hipDeviceSynchronize();

            if (stat != HIPBLAS_STATUS_SUCCESS) {
                printf("stat = %s\n", _cudaGetErrorEnum(stat));
                hipFree(dev_A);
                hipFree(dev_Q);
                hipFree(dev_R);
                hipFree(dev_x);
                //cudaFree(dev_v);
                hipFree(dev_Rv);
                hipblasDestroy(handle);
                return EXIT_FAILURE;
            }

            // Sets R[u:m, u:(s+r)] = R[u:m, u:(s+r)] - beta * v @ dev_Rv.T
            printf("\n");
            printf("===== hipblasDger =====\n");

            printf("dev_Vmat[:, j] = \n");
            print_ptr<<<1,1>>>(dev_Vmat + j + j * R_rows, R_rows - j, 1);
            hipDeviceSynchronize();

            printf("dev_Rv + u\n");
            print_ptr<<<1,1>>>((dev_Rv + u), (s + r) - u, 1);
            hipDeviceSynchronize();

            printf("B[j] = %f\n", *(B + j));
            hipDeviceSynchronize();

            printf("rows = %d, cols = %d\n", R_rows - u, (s + r) - u);

            stat = hipblasDger(handle, 
                              R_rows - u, (s + r) - u,
                              (B + j),
                              //(dev_Vmat + u + u * R_rows), 1,
                              (dev_Vmat + j + j * R_rows), 1,
                              (dev_Rv + u), 1,
                              (dev_R + u + u * R_rows), R_rows);

            printf("R[u:m, u:(s + r)] = \n");
            print_ptr<<<1,1>>>(dev_R + u + u * R_rows, R_rows - u, (s + r) - u);
            hipDeviceSynchronize();

            if (stat != HIPBLAS_STATUS_SUCCESS) {
                printf("stat = %s\n", _cudaGetErrorEnum(stat));
                hipFree(dev_A);
                hipFree(dev_Q);
                hipFree(dev_R);
                hipFree(dev_x);
                //cudaFree(dev_v);
                hipFree(dev_Rv);
                hipblasDestroy(handle);
                return EXIT_FAILURE;
            }

            //printf("dev_B[j] = \n");
            //print_ptr_num<<<1,1>>>((dev_B + j));
            //cudaDeviceSynchronize();

            //printf("beta = %f\n", beta);
            //cudaDeviceSynchronize();
            printf("===== R at end of j loop =====\n");
            printf("R = \n");
            print_ptr<<<1,1>>>(dev_R, R_rows, R_cols);
            hipDeviceSynchronize();
        }

        // copy W, Y vectors

        //Y = np.atleast_2d(V[:,0]).T
        // copy first column of dev_Vmat into first column of Y
        printf("\n");
        printf("===== GENERATING W, Y =====\n");
        cudaStat = hipMemcpy(dev_Y, dev_Vmat, A_rows * sizeof(double), hipMemcpyDeviceToDevice); // Y = V[:, 0]
        cudaStat = hipMemcpy(dev_W, dev_Vmat, A_rows * sizeof(double), hipMemcpyDeviceToDevice); // W = V[:, 0]
        hipblasDscal(handle, A_rows, B, dev_W, 1); // W = -B[0] * V[:, 0]

        for (int i = 1; i < r; i++) {
            // gemv Yt_v <-- Y.T @ v
            printf("=====  i = %d  =====\n", i);
            printf("Y = \n");
            print_ptr<<<1,1>>>(dev_Y, A_rows, r);
            hipDeviceSynchronize();
            printf("v = \n");
            print_ptr<<<1,1>>>((dev_Vmat + i * A_rows), A_rows, 1);
            hipDeviceSynchronize();

            stat = hipblasDgemv(handle, HIPBLAS_OP_T, 
                               //A_rows, r,
                               A_rows, i,
                               &alpha, 
                               dev_Y, A_rows,
                               //(dev_Vmat + i * A_rows), A_rows,
                               (dev_Vmat + i * A_rows), 1,
                               &gamma,
                               dev_Yt_v, 1);

            printf("Yt_v = \n");
            print_ptr<<<1,1>>>(dev_Yt_v , r, 1);
            hipDeviceSynchronize();

            printf("W = \n");
            print_ptr<<<1,1>>>(dev_W, A_rows, r);
            hipDeviceSynchronize();

            // gemm WYt_v <-- -B[j] * W @ Yt_v
            stat = hipblasDgemv(handle, HIPBLAS_OP_N, 
                               A_rows, i,
                               (B + i), 
                               dev_W, A_rows,
                               dev_Yt_v, 1,
                               &gamma,
                               dev_WYt_v, 1);

            printf("WYt_v = \n");
            print_ptr<<<1,1>>>(dev_WYt_v, A_rows, 1);
            hipDeviceSynchronize();

            // axpy WYt_v <-- -B[j]v + WYt_v
            stat = hipblasDaxpy(handle, A_rows, 
                               (B + i),
                               dev_Vmat + i * A_rows, 1,
                               dev_WYt_v, 1);

            // scal WYt_v <-- -B[j] * WYt_v
            //cublasDscal(handle, A_rows, (B + i), dev_WYt_v, 1);

            printf("z = \n");
            print_ptr<<<1,1>>>(dev_WYt_v, A_rows, 1);
            hipDeviceSynchronize();

            // memcpy Y + r*A_rows <-- v
            cudaStat = hipMemcpy(dev_Y + i * A_rows, dev_Vmat + i * A_rows, A_rows * sizeof(double), hipMemcpyDeviceToDevice);
            // memcpy W + r*A_rows <-- z
            cudaStat = hipMemcpy(dev_W + i * A_rows, dev_WYt_v, A_rows * sizeof(double), hipMemcpyDeviceToDevice);

            printf("Y = \n");
            print_ptr<<<1,1>>>(dev_Y, A_rows, r);
            hipDeviceSynchronize();

            printf("W = \n");
            print_ptr<<<1,1>>>(dev_W, A_rows, r);
            hipDeviceSynchronize();
        }

        printf("===== COMPLETED W, Y UPDATE =====\n");

        printf("W = \n");
        print_ptr<<<1,1>>>(dev_W, A_rows, r);
        hipDeviceSynchronize();
        printf("Y = \n");
        print_ptr<<<1,1>>>(dev_Y, A_rows, r);
        hipDeviceSynchronize();

        printf("===== UPDATING Q,R =====\n");

        // Update Q, R
        // Update R
        // R[s:, s + r:] = R[s:, s + r:] + Y @ (W.T @ R[s:, s + r:])
        // W.T is (r x R_rows). R[s: s + r:] is (R_rows - s) x (R_cols - (s + r))
        // gemm  dev_WTR <-- W.T @ (R + s + (s + r) * R_rows)
        stat = hipblasDgemm(handle, 
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           r, R_cols - (s + r), R_rows - s,
                           &alpha,
                           dev_W, R_rows,
                           (dev_R + s + (s + r) * R_rows), R_rows,
                           &gamma,
                           dev_WTR, r);
        // gemm  dev_YWTR <-- Y @ dev_WTR
        // Y is A_rows x r
        // dev_WTR is r x (R_cols - (s + r))
        stat = hipblasDgemm(handle, 
                           HIPBLAS_OP_N, HIPBLAS_OP_N,
                           A_rows, R_cols - (s + r), r,
                           &alpha, 
                           dev_Y, A_rows,
                           dev_WTR, r,
                           &gamma,
                           dev_YWTR, A_rows);

        // axpy  R + s + (s + r) * R_rows <-- dev_YWTR + (R + s + (s + r) * R_rows)
        stat = hipblasDaxpy(handle, A_rows * (R_cols - (s + r)), &alpha, dev_YWTR, 1, (dev_R + s + (s + r) * R_rows), 1);

        printf("===== FINAL R AFTER BLOCK k = %d =====\n", k);
        print_ptr<<<1,1>>>(dev_R, R_rows, R_cols);
        hipDeviceSynchronize();

        // Update Q
        // Q[:, s:] = Q[:, s:] + Q[:, s:] @ W  @ Y.T
        // gemm dev_WYT <-- W @ Y.T
        // W is (A_rows - s ) x r, Y.T is r x (A_rows - s)
        printf("first dgemm\n");
        stat = hipblasDgemm(handle,
                           HIPBLAS_OP_N, HIPBLAS_OP_T,
                           R_rows - s, R_rows - s, r,
                           &alpha,
                           dev_W, R_rows,
                           dev_Y, R_rows,
                           &gamma,
                           dev_WYT, R_rows);

        printf("R_rows - s = %d\n", R_rows - s);
        printf("r = %d\n", r);
        printf("dev_W = %p\n", dev_W);
        printf("R_rows= %d\n", R_rows);
        printf("dev_Y = %p\n", dev_Y);
        printf("r = %d\n", r);
        printf("gamma = %f\n", gamma);
        printf("dev_WYT = %p\n", dev_WYT);
        printf("R_rows = %d\n", R_rows);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf("stat = %s\n", _cudaGetErrorEnum(stat));
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        } 
        printf("dev_WYT\n");
        print_ptr<<<1,1>>>(dev_WYT, R_rows - s, R_rows - s);
        hipDeviceSynchronize();

        // gemm dev_QWYT <-- (Q + s * Q_rows) @ dev_WYT
        // Q[:, s:] is A_rows x (A_rows - s)
        // dev_WYT is (A_rows - s) x (A_rows - s)
        printf("second dgemm\n");
        stat = hipblasDgemm(handle,
                           HIPBLAS_OP_N, HIPBLAS_OP_N,
                           A_rows, A_rows - s, A_rows - s,
                           &alpha,
                           dev_Q + s * Q_rows, Q_rows,
                           dev_WYT, A_rows,
                           &gamma,
                           dev_QWYT, A_rows);
        // axpy (Q + s * Q_rows) <-- dev_QWYT + (Q + s * Q_rows)
        stat = hipblasDaxpy(handle, Q_rows * (Q_cols - s), &alpha, dev_QWYT, 1, (dev_Q + s * Q_rows), 1);

        printf("===== FINAL Q AFTER BLOCK k = %d =====\n", k);
        print_ptr<<<1,1>>>(dev_Q, Q_rows, Q_cols);
        hipDeviceSynchronize();

        // R[s:, s + r:] is,  at max, R_rows x R_cols - r
    }

    cudaStat = hipMemcpy(A, dev_A, A_rows * A_cols * sizeof(double), hipMemcpyDeviceToHost);
    cudaStat = hipMemcpy(Q, dev_Q, Q_rows * Q_cols * sizeof(double), hipMemcpyDeviceToHost);
    cudaStat = hipMemcpy(R, dev_R, R_rows * R_cols * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dev_A);
    hipFree(dev_Q);
    hipFree(dev_R);
    hipFree(dev_x);
    //cudaFree(dev_v);
    hipFree(dev_Rv);
    hipblasDestroy(handle);
    return status;
}
