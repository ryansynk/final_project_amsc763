#ifdef DEBUG
#define DEBUG_TEST 1
#else
#define DEBUG_TEST 0
#endif

#define debug_print(fmt, ...) \
            do { if (DEBUG_TEST) printf(fmt, __VA_ARGS__); } while (0)

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

__global__ void print_ptr(double *A, int A_rows, int A_cols, int lda) {
    // Data stored in column-major order
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < A_cols; j++) {
            if (A[j*lda + i] >= 0) {
                debug_print(" %f ", A[j*lda + i]);
            } else {
                debug_print("%f ", A[j*lda + i]);
            }
        }
        debug_print("%s", "\n");
    }
}

__global__ void print_ptr_num(double *A) {
    // Data stored in column-major order
    debug_print("%f\n", *A);
}

static const char *_cudaGetErrorEnum(hipblasStatus_t error) {
    switch (error) {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
    }
    return "<unknown>";
}

// On-device householder transform
//__global__ void gpu_house(cublasHandle_t handle, int row_beg, int rows, int cols, double *d_x, double *d_v, double *beta, int col) {
//TODO get rid of cols parameter
__global__ void gpu_house(hipblasHandle_t handle, int k, int rows, int cols, double *d_x, double *d_v, double *beta) {
    if (d_v == NULL || d_x == NULL || beta == NULL) {
        return;
    }
    double norm_x = 0.0;
    *beta = 0.0;
    // Data stored in column-major order
    // d_v gets first column
    for (int i = 0; i < rows - k; i++) {
        (d_v + k)[i] = (d_x + k + k * rows)[i];
        norm_x += (d_x + k + k * rows)[i] * (d_x + k + k * rows)[i];
    }

    norm_x = sqrt(norm_x);
    if (norm_x == 0.0) {
        *beta = 0.0;
    } else {
        (d_v + k)[0] = (d_v + k)[0] - norm_x;

        for (int i = 0; i < rows - k; i++) {
            *beta += (d_v + k)[i] * (d_v + k)[i];
        }
        *beta = -2.0 / *beta;
    }
}

// extern "C"
//int gpu_qr(Matrix *A, Matrix *Q, Matrix *R, int A_rows, int A_cols) {
extern "C" int gpu_qr(double *A, double *Q, double *R, int A_rows, int A_cols) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int status = EXIT_SUCCESS;
    if (A == NULL || Q == NULL || R == NULL) {
        status = EXIT_FAILURE;
    } 

    // Initialize Q to identity
    int Q_rows = A_rows;
    int Q_cols = A_rows;
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < A_rows; j++) {
            if (i == j) {
                Q[i * A_rows + j] = 1.0;
            } else {
                Q[i * A_rows + j] = 0.0;
            }
        }
    }

    // Initialize R to A
    int R_rows = A_rows;
    int R_cols = A_cols;
    for (int i = 0; i < A_rows * A_cols; i++) {
        R[i] = A[i];
    }

    // Cuda/CUBLAS initializations
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        debug_print("stat = %s\n", _cudaGetErrorEnum(stat));
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    double* dev_A = NULL;
    double* dev_Q = NULL;
    double* dev_R = NULL;
    double* dev_x = NULL;
    double* dev_v = NULL;
    double* dev_Rv = NULL;
    double* dev_Qv = NULL;
    double* dev_beta = NULL;
    double* dev_alpha = NULL;
    double* dev_gamma = NULL;

    double beta = 0.0;
    double alpha = 1.0;
    double gamma = 0.0;

    cudaStat = hipMalloc(&dev_A, A_rows * A_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_Q, Q_rows * Q_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_R, R_rows * R_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_x, R_rows * sizeof(double));
    cudaStat = hipMalloc(&dev_v, R_rows * sizeof(double));
    cudaStat = hipMalloc(&dev_Rv, R_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_Qv, Q_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_beta, sizeof(double));
    cudaStat = hipMalloc(&dev_alpha, sizeof(double));
    cudaStat = hipMalloc(&dev_gamma, sizeof(double));

    if (cudaStat != hipSuccess) {
        debug_print("%s\n", "device memory allocation failed");
        return EXIT_FAILURE;
    }

    cudaStat = hipMemcpy(dev_A, A, A_rows * A_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_Q, Q, Q_rows * Q_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_R, R, R_rows * R_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_alpha, &alpha, sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_gamma, &gamma, sizeof(double), hipMemcpyHostToDevice);

    if (cudaStat != hipSuccess) {
        debug_print("%s\n", "host to device memory copy failed");
        return EXIT_FAILURE;
    }
    for (int k = 0; k < A_cols; k++) {
        gpu_house<<<1,1>>>(handle, k, R_rows, R_cols, dev_R, dev_v, dev_beta);
        cudaStat = hipMemcpy(&beta, dev_beta, sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        // Gets dev_Rv = R[k:m, k:n] @ v
        stat = hipblasDgemv(handle, HIPBLAS_OP_T, 
                           R_rows - k, R_cols - k, 
                           &alpha, 
                           (dev_R + k + k * R_rows), R_rows,
                           (dev_v + k), 1,
                           &gamma,
                           (dev_Rv + k), 1);

        if (stat != HIPBLAS_STATUS_SUCCESS) {
            debug_print("stat = %s\n", _cudaGetErrorEnum(stat));
            hipFree(dev_A);
            hipFree(dev_Q);
            hipFree(dev_R);
            hipFree(dev_x);
            hipFree(dev_v);
            hipFree(dev_Rv);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }

        // Sets R[k:m, k:n] = R[k:m, k:n] - beta * v @ dev_Rv.T
        stat = hipblasDger(handle, 
                          R_rows - k, R_cols - k,
                          &beta,
                          (dev_v + k), 1,
                          (dev_Rv + k), 1,
                          (dev_R + k + k * R_rows), R_rows);

        if (stat != HIPBLAS_STATUS_SUCCESS) {
            debug_print("stat = %s\n", _cudaGetErrorEnum(stat));
            hipFree(dev_A);
            hipFree(dev_Q);
            hipFree(dev_R);
            hipFree(dev_x);
            hipFree(dev_v);
            hipFree(dev_Rv);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }

        // Gets dev_Qv = Q[:, k:m] @ v
        stat = hipblasDgemv(handle, HIPBLAS_OP_N, 
                           Q_rows, Q_cols - k, 
                           &alpha, 
                           (dev_Q + k * Q_rows), Q_rows,
                           (dev_v + k), 1,
                           &gamma,
                           (dev_Qv + k), 1);

        if (stat != HIPBLAS_STATUS_SUCCESS) {
            debug_print("stat = %s\n", _cudaGetErrorEnum(stat));
            hipFree(dev_A);
            hipFree(dev_Q);
            hipFree(dev_R);
            hipFree(dev_x);
            hipFree(dev_v);
            hipFree(dev_Rv);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }
        debug_print("beta = %f\n", beta);

        // Sets Q[:, k:m] = Q[:, k:m] - beta * (Q[:, k:m] @ v) @ v.T
        stat = hipblasDger(handle, 
                          Q_rows, Q_cols - k,
                          &beta,
                          (dev_Qv + k), 1,
                          (dev_v + k), 1,
                          (dev_Q + k * Q_rows), Q_rows);

        if (stat != HIPBLAS_STATUS_SUCCESS) {
            debug_print("stat = %s\n", _cudaGetErrorEnum(stat));
            hipFree(dev_A);
            hipFree(dev_Q);
            hipFree(dev_R);
            hipFree(dev_x);
            hipFree(dev_v);
            hipFree(dev_Rv);
            hipblasDestroy(handle);
            return EXIT_FAILURE;
        }
    }

    cudaStat = hipMemcpy(A, dev_A, A_rows * A_cols * sizeof(double), hipMemcpyDeviceToHost);
    cudaStat = hipMemcpy(Q, dev_Q, Q_rows * Q_cols * sizeof(double), hipMemcpyDeviceToHost);
    cudaStat = hipMemcpy(R, dev_R, R_rows * R_cols * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dev_A);
    hipFree(dev_Q);
    hipFree(dev_R);
    hipFree(dev_x);
    hipFree(dev_v);
    hipFree(dev_Rv);
    hipblasDestroy(handle);
    return status;
}


extern "C" int gpu_block_qr(double *A, double *Q, double *R, int A_rows, int A_cols, int r, float *time) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int status = EXIT_SUCCESS;
    if (A == NULL || Q == NULL || R == NULL) {
        status = EXIT_FAILURE;
    } 

    // Initialize Q to identity
    int Q_rows = A_rows;
    int Q_cols = A_rows;
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < A_rows; j++) {
            if (i == j) {
                Q[i * A_rows + j] = 1.0;
            } else {
                Q[i * A_rows + j] = 0.0;
            }
        }
    }

    // Initialize R to Q
    int R_rows = A_rows;
    int R_cols = A_cols;
    for (int i = 0; i < A_rows * A_cols; i++) {
        R[i] = A[i];
    }

    // CuBLAS declarations
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        debug_print("stat = %s\n", _cudaGetErrorEnum(stat));
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    } 

    // Pointer declarations for matrices in computation
    double *dev_A, *dev_Q, *dev_R, *dev_x; 
    double *dev_Rv, *dev_Qv; //tmp values, R times v, Q times v
    double *dev_beta, *dev_alpha, *dev_gamma;
    double *dev_Y, *dev_W, *dev_B, *dev_Vmat, *B;
    double *dev_Yt_v, *dev_WYt_v;

    // intermediate -- used for updating R
    // W transpose times R and Y times W transpose times R
    double *dev_WTR,* dev_YWTR;

    // intermediate -- used for updating Q
    // "W times Y transpose" and "Q times W times Y transpose"
    double *dev_WYT, *dev_QWYT;

    double alpha = 1.0;
    double gamma = 0.0;
    int s = 0;
    int u = 0;

    // Allocating device memory
    cudaStat = hipMalloc(&dev_A, A_rows * A_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_Q, Q_rows * Q_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_R, R_rows * R_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_x, R_rows * sizeof(double));

    cudaStat = hipMalloc(&dev_Rv, R_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_Qv, Q_cols * sizeof(double));
    cudaStat = hipMalloc(&dev_beta, sizeof(double));
    cudaStat = hipMalloc(&dev_alpha, sizeof(double));
    cudaStat = hipMalloc(&dev_gamma, sizeof(double));

    cudaStat = hipMalloc(&dev_B, r * sizeof(double));
    cudaStat = hipMalloc(&dev_Vmat, A_rows * r * sizeof(double));
    hipMemset(dev_Vmat, 0, A_rows * r * sizeof(double));

    cudaStat = hipMalloc(&dev_Y, A_rows * r * sizeof(double));
    cudaStat = hipMalloc(&dev_W, A_rows * r * sizeof(double));

    cudaStat = hipMalloc(&dev_Yt_v, r * sizeof(double));
    cudaStat = hipMalloc(&dev_WYt_v, A_rows * sizeof(double));

    cudaStat = hipMalloc(&dev_WTR, r * (R_cols - r) * sizeof(double));
    cudaStat = hipMalloc(&dev_YWTR, R_rows * (R_cols - r) * sizeof(double));

    cudaStat = hipMalloc(&dev_WYT, R_rows * R_rows * sizeof(double));
    cudaStat = hipMalloc(&dev_QWYT, Q_rows * R_rows * sizeof(double));

    // Holds beta values
    B = (double *)malloc(r * sizeof(double));

    if (cudaStat != hipSuccess) {
        debug_print("%s\n", "device memory allocation failed");
        return EXIT_FAILURE;
    }

    // Copying A, Q, R, and constants to device memory
    cudaStat = hipMemcpy(dev_A, A, A_rows * A_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_Q, Q, Q_rows * Q_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_R, R, R_rows * R_cols * sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_alpha, &alpha, sizeof(double), hipMemcpyHostToDevice);
    cudaStat = hipMemcpy(dev_gamma, &gamma, sizeof(double), hipMemcpyHostToDevice);

    if (cudaStat != hipSuccess) {
        debug_print("%s\n", "host to device memory copy failed");
        return EXIT_FAILURE;
    }

    hipEventRecord(start);

    for (int k = 0; k < (A_cols / r); k++) {
        debug_print("k = %d\n", k);
        s = k * r;
        //TODO: zero out dev_Vmat?
        for (int j = 0; j < r; j++) {
            u = s + j;

            gpu_house<<<1,1>>>(handle, u, R_rows, R_cols, dev_R, (dev_Vmat + j * R_rows + j - u), dev_beta);
            cudaStat = hipMemcpy((B + j), dev_beta, sizeof(double), hipMemcpyDeviceToHost);

            stat = hipblasDgemv(handle, HIPBLAS_OP_T, 
                               R_rows - u, (s + r) - u, 
                               &alpha, 
                               (dev_R + u + u * R_rows), R_rows,
                               (dev_Vmat + j + j * R_rows), 1,
                               &gamma,
                               (dev_Rv + u), 1);


            // Sets R[u:m, u:(s+r)] = R[u:m, u:(s+r)] - beta * v @ dev_Rv.T
            stat = hipblasDger(handle, 
                              R_rows - u, (s + r) - u,
                              (B + j),
                              (dev_Vmat + j + j * R_rows), 1,
                              (dev_Rv + u), 1,
                              (dev_R + u + u * R_rows), R_rows);

        }

        // copy W, Y vectors

        // copy first column of dev_Vmat into first column of Y
        cudaStat = hipMemcpy(dev_Y, dev_Vmat, (A_rows - s) * sizeof(double), hipMemcpyDeviceToDevice); // Y = V[:, 0]
        cudaStat = hipMemcpy(dev_W, dev_Vmat, (A_rows - s) * sizeof(double), hipMemcpyDeviceToDevice); // W = V[:, 0]
        hipblasDscal(handle, (A_rows - s), B, dev_W, 1); // W = -B[0] * V[:, 0]

        for (int i = 1; i < r; i++) {
            // gemv Yt_v <-- Y.T @ v
            // Y is (A_rows - s) ==> Yt_v is A_rows - s
            stat = hipblasDgemv(handle, HIPBLAS_OP_T, 
                               A_rows - s, i,
                               &alpha, 
                               dev_Y, A_rows,
                               (dev_Vmat + i * A_rows), 1,
                               &gamma,
                               dev_Yt_v, 1);

            // gemm WYt_v <-- -B[j] * W @ Yt_v
            stat = hipblasDgemv(handle, HIPBLAS_OP_N, 
                               A_rows - s, i,
                               (B + i), 
                               dev_W, A_rows,
                               dev_Yt_v, 1,
                               &gamma,
                               dev_WYt_v, 1);

            // axpy WYt_v <-- -B[j]v + WYt_v
            stat = hipblasDaxpy(handle, A_rows - s, 
                               (B + i),
                               dev_Vmat + i * A_rows, 1,
                               dev_WYt_v, 1);

            // memcpy Y + r*A_rows <-- v
            cudaStat = hipMemcpy(dev_Y + i * A_rows, 
                                  dev_Vmat + i * A_rows, 
                                  A_rows * sizeof(double), 
                                  hipMemcpyDeviceToDevice);
            // memcpy W + r*A_rows <-- z
            cudaStat = hipMemcpy(dev_W + i * A_rows, 
                                  dev_WYt_v, 
                                  A_rows * sizeof(double), 
                                  hipMemcpyDeviceToDevice);
        }

        // Update Q, R

        // Update R
        // R[s:, s + r:] = R[s:, s + r:] + Y @ (W.T @ R[s:, s + r:])
        // gemm  dev_WTR <-- W.T @ (R + s + (s + r) * R_rows)
        stat = hipblasDgemm(handle, 
                           HIPBLAS_OP_T, HIPBLAS_OP_N,
                           r, R_cols - (s + r), R_rows - s,
                           &alpha,
                           dev_W, R_rows,
                           (dev_R + s + (s + r) * R_rows), R_rows,
                           &gamma,
                           dev_WTR, r);

        // gemm  dev_YWTR <-- Y @ dev_WTR
        // Y is (A_rows - s )x r
        // dev_WTR is r x (R_cols - (s + r))
        stat = hipblasDgemm(handle, 
                           HIPBLAS_OP_N, HIPBLAS_OP_N,
                           A_rows - s, R_cols - (s + r), r,
                           &alpha, 
                           dev_Y, A_rows,
                           dev_WTR, r,
                           &gamma,
                           dev_YWTR, A_rows);

        // iterate over each column of submatrix, and axpy 
        // TODO: This can be paralellized?
        for (int i = 0; i < R_cols - (s + r); i++) {
            stat = hipblasDaxpy(handle, (A_rows - s), &alpha, dev_YWTR + i * (A_rows), 1, (dev_R + s + (s + r) * R_rows + i * R_rows), 1);
        }

        // Update Q
        // Q[:, s:] = Q[:, s:] + Q[:, s:] @ W  @ Y.T
        // gemm dev_WYT <-- W @ Y.T
        stat = hipblasDgemm(handle,
                           HIPBLAS_OP_N, HIPBLAS_OP_T,
                           R_rows - s, R_rows - s, r,
                           &alpha,
                           dev_W, R_rows,
                           dev_Y, R_rows,
                           &gamma,
                           dev_WYT, R_rows);

        // gemm dev_QWYT <-- (Q + s * Q_rows) @ dev_WYT
        // Q[:, s:] is A_rows x (A_rows - s)
        // dev_WYT is (A_rows - s) x (A_rows - s)
        stat = hipblasDgemm(handle,
                           HIPBLAS_OP_N, HIPBLAS_OP_N,
                           A_rows, A_rows - s, A_rows - s,
                           &alpha,
                           dev_Q + s * Q_rows, Q_rows,
                           dev_WYT, A_rows,
                           &gamma,
                           dev_QWYT, A_rows);

        // axpy (Q + s * Q_rows) <-- dev_QWYT + (Q + s * Q_rows)
        stat = hipblasDaxpy(handle, Q_rows * (Q_cols - s), &alpha, dev_QWYT, 1, (dev_Q + s * Q_rows), 1);
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    cudaStat = hipMemcpy(A, dev_A, A_rows * A_cols * sizeof(double), hipMemcpyDeviceToHost);
    cudaStat = hipMemcpy(Q, dev_Q, Q_rows * Q_cols * sizeof(double), hipMemcpyDeviceToHost);
    cudaStat = hipMemcpy(R, dev_R, R_rows * R_cols * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dev_A);
    hipFree(dev_Q);
    hipFree(dev_R);
    hipFree(dev_x);
    hipFree(dev_Rv);
    hipblasDestroy(handle);

    float milliseconds = 0.0;
    hipEventElapsedTime(&milliseconds, start, stop);
    *time = (milliseconds / 1000.0);
    return status;
}
